#include "hip/hip_runtime.h"
#include "udist.cuh"
#include "random.cuh"

__global__
/**
 * @brief CUDA kernel for setting none(frozen) u distribution
 * 
 * @param d_tiles 
 * @param d_u 
 */
void _set_none( t_part_tile const * const __restrict__ d_tiles,
    float3 * const __restrict__ d_u ) {

    const int tid = blockIdx.y * gridDim.x + blockIdx.x;

    const int offset = d_tiles[tid].pos;
    const int np     = d_tiles[tid].n;
    float3 __restrict__ * const u  = &d_u[ offset ];

    for( int i = threadIdx.x; i < np; i+= blockDim.x ) {
        u[i] = make_float3(0,0,0);
    }
}

/**
 * @brief Sets none(0 temperature, 0 fluid) u distribution
 * 
 * @param part  Particle data
 */
void UDistribution::None::set( Particles & part ) const {

    dim3 grid( part.ntiles.x, part.ntiles.y );
    dim3 block( 64 );
    
    _set_none <<< grid, block >>> ( part.tiles, part.u );
}

__global__
/**
 * @brief CUDA kernel for setting cold u distribution
 * 
 * @param d_tiles 
 * @param d_u 
 * @param ufl 
 */
void _set_cold( t_part_tile const * const __restrict__ d_tiles,
    float3 * const __restrict__ d_u, float3 const ufl ) {

    const int tid = blockIdx.y * gridDim.x + blockIdx.x;

    const int offset = d_tiles[tid].pos;
    const int np     = d_tiles[tid].n;
    float3 __restrict__ * const u  = &d_u[ offset ];

    for( int i = threadIdx.x; i < np; i+= blockDim.x ) {
        u[i] = ufl;
    }
}

/**
 * @brief Sets cold(0 temperatures) u distribution
 * 
 * @param part  Particle data
 */
void UDistribution::Cold::set( Particles & part ) const {

    dim3 grid( part.ntiles.x, part.ntiles.y );
    dim3 block( 64 );
    
    _set_none <<< grid, block >>> ( part.tiles, part.u );
}


__global__
/**
 * @brief Sets particle momentum
 * 
 * @param d_tile    Tile information
 * @param d_u       Particle buffer (momenta)
 * @param seed      Seed for random number generator
 * @param uth       Thermal distribution width
 * @param ufl       Fluid momentum
 */
void _set_thermal( 
    t_part_tile const * const __restrict__ d_tiles,
    float3 * const __restrict__ d_u, 
    const uint2 seed, const float3 uth, const float3 ufl ) {

    // Tile ID
    const int tid = blockIdx.y * gridDim.x + blockIdx.x;

    // Initialize random state variables
    uint2 state;
    double norm;
    rand_init( seed, state, norm );

    // Set particle momenta
    const int offset = d_tiles[tid].pos;
    const int np     = d_tiles[tid].n;
    float3 __restrict__ * const u  = &d_u[ offset ];

    for( int i = threadIdx.x; i < np; i+= blockDim.x ) {
        u[i] = make_float3(
            ufl.x + uth.x * rand_norm( state, norm ),
            ufl.y + uth.y * rand_norm( state, norm ),
            ufl.z + uth.z * rand_norm( state, norm )
        );
    }
}

/**
 * @brief Sets momentum of all particles in object using uth / ufl
 * 
 */
void UDistribution::Thermal::set( Particles & part ) const {

    std::cout << "(*info*) Setting 'Thermal' u distribution\n";
    std::cout << "(*info*) uth(" << uth.x << "," << uth.y << "," << uth.z << ")\n";
    std::cout << "(*info*) ufl(" << ufl.x << "," << ufl.y << "," << ufl.z << ")\n";

    // Set thermal momentum
    dim3 grid( part.ntiles.x, part.ntiles.y );
    dim3 block( 64 );
    
    uint2 seed = {12345, 67890};
    _set_thermal <<< grid, block >>> ( 
        part.tiles, part.u, seed, uth, ufl
    );
}

__global__
/**
 * @brief Sets particle momentum correcting local ufl fluctuations
 * 
 * @param d_tile    Tile information
 * @param d_u       Particle buffer (momenta)
 * @param d_ix      Particle buffer (momenta)
 * @param nx        Tile size
 * @param seed      Seed for random number generator
 * @param uth       Thermal distribution width
 * @param ufl       Fluid momentum
 * @param npmin     Minimum number of particles in cell to apply correction
 */
void _set_thermal_corr( 
    t_part_tile const * const __restrict__ d_tiles,
    float3 * const __restrict__ d_u, int2 const * const __restrict__ d_ix, uint2 const nx, 
    uint2 const seed, float3 const uth, float3 const ufl, int const npmin ) {

    auto block = cg::this_thread_block();

    extern __shared__ char buffer[];
    int * const __restrict__ npcell = (int*) buffer;
    float3 * const __restrict__ fluid = (float3*) (buffer + nx.x*nx.y*sizeof(int));

    for( int idx = threadIdx.x; idx < nx.x*nx.y; idx += blockDim.x ) {
        npcell[idx] = 0;
        fluid[idx].x = 0;
        fluid[idx].y = 0;
        fluid[idx].z = 0;
    }

    block.sync();

    // Tile ID
    const int tid = blockIdx.y * gridDim.x + blockIdx.x;

    // Initialize random state variables
    uint2 state;
    double norm;
    rand_init( seed, state, norm );

    // Set particle momenta
    const int offset = d_tiles[tid].pos;
    const int np     = d_tiles[tid].n;
    float3 * const __restrict__ u  = &d_u[ offset ];
    int2 const * const __restrict__ ix = &d_ix[offset];

    for( int i = threadIdx.x; i < np; i+= blockDim.x ) {
        float3 upart = make_float3(
            uth.x * rand_norm( state, norm ),
            uth.y * rand_norm( state, norm ),
            uth.z * rand_norm( state, norm )
        );
        u[i] = upart;

        int const idx = ix[i].x + nx.x * ix[i].y;

        atomicAdd( &npcell[ idx ], 1 );
        atomicAdd( &fluid[ idx ].x, upart.x );
        atomicAdd( &fluid[ idx ].y, upart.y );
        atomicAdd( &fluid[ idx ].z, upart.z );
    }

    block.sync();

    for( int idx = threadIdx.x; idx < nx.x*nx.y; idx+= blockDim.x ) {
        if ( npcell[idx] > npmin ) {
            fluid[idx].x /= npcell[idx];
            fluid[idx].y /= npcell[idx];
            fluid[idx].z /= npcell[idx];
        } else {
            fluid[idx] = make_float3(0,0,0);
        }
    }

    block.sync();

    for( int i = threadIdx.x; i < np; i+= blockDim.x ) {
        float3 upart = u[i];
        int const idx = ix[i].x + nx.x * ix[i].y;
        
        upart.x += ufl.x - fluid[idx].x;
        upart.y += ufl.y - fluid[idx].y;
        upart.z += ufl.z - fluid[idx].z;

        u[i] = upart;
    }
}

/**
 * @brief Sets particle momentum correcting local ufl fluctuations
 * 
 */
void UDistribution::ThermalCorr::set( Particles & part ) const {

    // Set thermal momentum
    dim3 grid( part.ntiles.x, part.ntiles.y );
    dim3 block( 64 );

    size_t shm_size = part.nx.x * part.nx.y * (sizeof(float3) + sizeof(int));
    
    uint2 seed = {12345, 67890};
    _set_thermal_corr <<< grid, block, shm_size >>> ( 
        part.tiles, part.u, part.ix, part.nx, seed, uth, ufl, npmin
    );
}