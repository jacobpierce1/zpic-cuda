#include "hip/hip_runtime.h"
#include "particles.cuh"
#include <iostream>
#include <string>

#include "util.cuh"

#include "timer.cuh"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg=cooperative_groups;


__global__
void _init_tiles_kernel( 
    int * const __restrict__ d_tile_offset, 
    int * const __restrict__ d_tile_np, 
    int * const __restrict__ d_tile_np2, 
    unsigned int const max_np_tile ) {

    const int i = blockIdx.y * gridDim.x + blockIdx.x;

    d_tile_offset[i] = i * max_np_tile;
    d_tile_np[i]  = 0;
    d_tile_np2[i] = 0;
}

/**
 * @brief Construct a new Particle Buffer:: Particle Buffer object
 * 
 * @param ntiles        Number of tiles (x,y)
 * @param nx            Tile size
 * @param max_np_tile   Maximum number of particles per tile
 */
__host__
Particles::Particles(uint2 const ntiles, uint2 const nx, unsigned int const max_np_tile ) :
    ntiles( ntiles ), nx( nx ), max_np_tile( max_np_tile ), periodic( make_int2(1,1) )
{    
    size_t size = ntiles.x * ntiles.y * max_np_tile;
    malloc_dev( ix, size );
    malloc_dev( x, size );
    malloc_dev( u, size );

    malloc_dev( idx, size );

    // Allocate tile information array on device and initialize using a CUDA kernel

    malloc_dev( tile_offset, ntiles.x * ntiles.y );
    malloc_dev( tile_np, ntiles.x * ntiles.y );
    malloc_dev( tile_np2, ntiles.x * ntiles.y );

    dim3 grid( ntiles.x, ntiles.y );
    _init_tiles_kernel <<< grid, 1 >>> ( tile_offset, tile_np, tile_np2, max_np_tile );
};


__global__
/**
 * @brief CUDA Kernel for getting total number of particles
 * 
 * Note that the kernel does not reset the output total value
 * 
 * @param d_tiles   Tile information
 * @param ntiles    total number of tiles
 * @param total     (out) total number of particles
 */
void _np_kernel( int const * const __restrict__ d_tile_np, 
    unsigned int const ntiles, unsigned int * const __restrict__ total) {
    auto group = cg::this_thread_block();
    auto warp  = cg::tiled_partition<32>(group);

    unsigned int np = 0;
    for( int i = group.thread_rank(); i < ntiles; i += group.num_threads() )
        np += d_tile_np[i];
    
    np = cg::reduce( warp, np, cg::plus<unsigned int>());
    if ( warp.thread_rank() == 0 ) atomicAdd( total, np );
}

__host__
/**
 * @brief Gets total number of particles on device
 * 
 * @return unsigned long long   Total number of particles
 */
unsigned int Particles::np() {
    _dev_tmp_uint = 0;
    auto size = ntiles.x*ntiles.y;
    auto block = ( size < 1024 ) ? size : 1024 ;
    auto grid = (size-1)/block + 1;
    _np_kernel <<< grid, block >>> ( tile_np, size, _dev_tmp_uint.ptr() );
    return _dev_tmp_uint.get();
}

__global__
void _np_max_tile( int const * const __restrict__ d_tile_np, 
    unsigned int const ntiles, unsigned int * const __restrict__ max) {
    auto group = cg::this_thread_block();
    auto warp  = cg::tiled_partition<32>(group);

    unsigned int v = 0;
    for( int i = group.thread_rank(); i < ntiles; i += group.num_threads() ) {
        int tile_np = d_tile_np[i];
        if ( tile_np > v ) v = tile_np;
    }
    
    v = cg::reduce( warp, v, cg::greater<unsigned int>());
    if ( warp.thread_rank() == 0 ) atomicMax( max, v );
}

__host__
/**
 * @brief Gets maximum number of particles per tile
 * 
 * @return unsigned int 
 */
unsigned int Particles::np_max_tile() {
    _dev_tmp_uint = 0;
    auto size = ntiles.x*ntiles.y;
    auto block = ( size < 1024 ) ? size : 1024 ;
    auto grid = (size-1)/block + 1;
    _np_max_tile <<< grid, block >>> ( tile_np, size, _dev_tmp_uint.ptr() );
    return _dev_tmp_uint.get();
}

__global__
void _np_min_tile( int const * const __restrict__ d_tile_np, 
    unsigned int const ntiles, unsigned int * const __restrict__ max) {
    auto group = cg::this_thread_block();
    auto warp  = cg::tiled_partition<32>(group);

    unsigned int v = 0;
    for( int i = group.thread_rank(); i < ntiles; i += group.num_threads() ) {
        int tile_np = d_tile_np[i];
        if ( tile_np > v ) v = tile_np;
    }
    
    v = cg::reduce( warp, v, cg::less<unsigned int>());
    if ( warp.thread_rank() == 0 ) atomicMin( max, v );
}

__host__
/**
 * @brief Gets minimum number of particles per tile
 * 
 * @return unsigned int 
 */
unsigned int Particles::np_min_tile() {
    _dev_tmp_uint = 0;
    auto size = ntiles.x*ntiles.y;
    auto block = ( size < 1024 ) ? size : 1024 ;
    auto grid = (size-1)/block + 1;
    _np_max_tile <<< grid, block >>> ( tile_np, size, _dev_tmp_uint.ptr() );
    return _dev_tmp_uint.get();
}

__global__
void _np_exscan_kernel( 
    unsigned int * const __restrict__ idx,
    int const * const __restrict__ d_tile_np, unsigned int const ntiles,
    unsigned int * const __restrict__ total) {

    __shared__ unsigned int tmp[ 32 ];
    __shared__ unsigned int prev;

    auto block = cg::this_thread_block();
    auto warp  = cg::tiled_partition<32>(block);

    prev = 0;

    for( unsigned int i = block.thread_rank(); i < ntiles; i += block.num_threads() ) {
        unsigned int s = d_tile_np[i];

        unsigned int v = cg::exclusive_scan( warp, s, cg::plus<unsigned int>());
        if ( warp.thread_rank() == warp.num_threads() - 1 ) tmp[ warp.meta_group_rank() ] = v + s;
        block.sync();

        if ( warp.meta_group_rank() == 0 ) {
            auto t = tmp[ warp.thread_rank() ];
            t = cg::exclusive_scan( warp, t, cg::plus<unsigned int>());
            tmp[ warp.thread_rank() ] = t + prev;
        }
        block.sync();

        v += tmp[ warp.meta_group_rank() ];
        idx[i] = v;

        if ((block.thread_rank() == block.num_threads() - 1) || ( i + 1 == ntiles ) )
            prev = v + s;
        block.sync();
    }

    if ( block.thread_rank() == 0 ) *total = prev;

}

/**
 * @brief Exclusive scan of number of particles per tile
 * 
 * This is used for compacting operations
 * 
 * @param d_offset          Output array on device, must be of size ntiles.x * ntiles.y
 * @return unsigned int     Total number of particles
 */
unsigned int Particles::np_exscan( unsigned int * __restrict__ d_offset ) {

    auto size = ntiles.x*ntiles.y;
    auto block = ( size < 1024 ) ? size : 1024 ;
    _dev_tmp_uint = 0;
    _np_exscan_kernel <<< 1, block >>> ( d_offset, tile_np, size, _dev_tmp_uint.ptr() );
    return _dev_tmp_uint.get();
}

/**
 * @brief CUDA kernel for gathering particle data
 * 
 * @tparam quant        Quantiy to gather
 * @param d_ix          Particle data (cells)
 * @param d_x           Particle data (positions)
 * @param d_u           Particle data (generalized velocity)
 * @param d_tiles       Particle tile information
 * @param tile_nx       Size of tile grid
 * @param d_out_offset  Output array offsets
 * @param d_data        Output data
 */
template < part::quant quant >
__global__
void _gather_quant( 
    int2 const * const __restrict__ d_ix, 
    float2 const * const __restrict__ d_x, 
    float3 const * const __restrict__ d_u, 
    int const * const __restrict__ d_tile_offset, 
    int const * const __restrict__ d_tile_np, 
    uint2 const tile_nx,
    unsigned int const * const __restrict__ d_out_offset, 
    float * const __restrict__ d_data )
{    
    const int tid = blockIdx.y * gridDim.x + blockIdx.x;

    const int offset = d_tile_offset[tid];
    const int np     = d_tile_np[tid];

    int2   __restrict__ const * const ix = &d_ix[ offset ];
    float2 __restrict__ const * const x  = &d_x[ offset ];
    float3 __restrict__ const * const u  = &d_u[ offset ];
    
    unsigned int const out_offset = d_out_offset[ tid ];

    for( int idx = threadIdx.x; idx < np; idx += blockDim.x ) {
        float val;
        if ( quant == part::x )  val = (blockIdx.x * tile_nx.x + ix[idx].x) + (0.5f + x[idx].x);
        if ( quant == part::y )  val = (blockIdx.y * tile_nx.y + ix[idx].y) + (0.5f + x[idx].y);
        if ( quant == part::ux ) val = u[idx].x;
        if ( quant == part::uy ) val = u[idx].y;
        if ( quant == part::uz ) val = u[idx].z;
        d_data[ out_offset + idx ] = val;
    }
};


__host__
/**
 * @brief Gather data from a specific particle quantity in a device buffer
 * 
 * @param quant         Quantity to gather
 * @param h_data        Output data host buffer, assumed to have size >= np
 * @param np            Number of particles
 * @param d_data_offset Data offset in output array for each tile
 */
void Particles::gather( part::quant quant, float * const __restrict__ h_data, 
        float * const __restrict__ d_data, 
        unsigned int const np, unsigned int const * const __restrict__ d_out_offset ) {

    if ( np > 0 ) {
        dim3 grid( ntiles.x, ntiles.y );
        dim3 block( 1024 );

        // Gather data on device
        switch (quant) {
        case part::x : 
            _gather_quant<part::x> <<<grid,block>>>( ix, x, u, tile_offset, tile_np, nx, d_out_offset, d_data );
            break;
        case part::y:
            _gather_quant<part::y> <<<grid,block>>>( ix, x, u, tile_offset, tile_np, nx, d_out_offset, d_data );
            break;
        case part::ux:
            _gather_quant<part::ux> <<<grid,block>>>( ix, x, u, tile_offset, tile_np, nx, d_out_offset, d_data );
            break;
        case part::uy:
            _gather_quant<part::uy> <<<grid,block>>>( ix, x, u, tile_offset, tile_np, nx, d_out_offset, d_data );
            break;
        case part::uz:
            _gather_quant<part::uz> <<<grid,block>>>( ix, x, u, tile_offset, tile_np, nx, d_out_offset, d_data );
            break;
        }

        // Copy to host
        devhost_memcpy( h_data, d_data, np );
    }
}

__host__
/**
 * @brief Gather data from a specific particle quantity in a device buffer
 * 
 * This version will first do an exscan on the number of particles per tile to
 * determine the data offset on the outout buffer for each tile and call the 
 * above version.
 * 
 * @param quant     Quantity to gather
 * @param h_data    Output data host buffer, assumed to have size >= np
 */
void Particles::gather( part::quant quant, float * const __restrict__ h_data ) {
        
    unsigned int * d_out_offset;
    malloc_dev( d_out_offset, ntiles.x * ntiles.y );
    unsigned int np = np_exscan( d_out_offset );
    
    if ( np > 0 ) {
        float * d_data;
        malloc_dev( d_data, np );
        gather( quant, h_data, d_data, np, d_out_offset );
        free_dev( d_data );
    }
    
    free_dev( d_out_offset );
}

__host__
/**
 * @brief Save particle data to disk
 * 
 * @param info  Particle metadata (name, labels, units, etc.). Information is used to set file name
 * @param iter  Iteration metadata
 * @param path  Path where to save the file
 */
void Particles::save( zdf::part_info &info, zdf::iteration &iter, std::string path ) {

    // Get number of particles and data offsets
    unsigned int * d_out_offset;
    malloc_dev( d_out_offset, ntiles.x * ntiles.y );
    unsigned int np = np_exscan( d_out_offset );
    info.np = np;

    // Open file
    zdf::file part_file;
    zdf::open_part_file( part_file, info, iter, path+"/"+info.name );

    // Gather and save each quantity
    float *h_data = nullptr, *d_data = nullptr;
    if( np > 0 ) {
        malloc_host( h_data, np );
        malloc_dev( d_data, np );
    }

    gather( part::quant::x, h_data, d_data, np, d_out_offset );
    zdf::add_quant_part_file( part_file, "x", h_data, np );

    gather( part::quant::y, h_data, d_data, np, d_out_offset );
    zdf::add_quant_part_file( part_file, "y", h_data, np );

    gather( part::quant::ux, h_data, d_data, np, d_out_offset );
    zdf::add_quant_part_file( part_file, "ux", h_data, np );

    gather( part::quant::uy, h_data, d_data, np, d_out_offset );
    zdf::add_quant_part_file( part_file, "uy", h_data, np );

    gather( part::quant::uz, h_data, d_data, np, d_out_offset );
    zdf::add_quant_part_file( part_file, "uz", h_data, np );

    // Close the file
    zdf::close_file( part_file );

    // Cleanup
    if ( np > 0 ) {
        free_dev( d_data );
        free_host( h_data );
    }
    free_dev( d_out_offset );
}

/**
 * @brief CUDA kernel for copying particles out of the tile to a temp buffer
 * 
 * @tparam dir          Direction to check `coord::x` or `coord::y`
 * @param lim           Tile size along chosen direction
 * @param d_tiles       Tile information (main buffer)
 * @param d_ix          Particle cells (main buffer)
 * @param d_x           Particle positions (main buffer)
 * @param d_u           Particle momenta (main buffer)
 * @param tmp_d_tiles   Tile information (temp buffer)
 * @param tmp_d_ix      Particle cells (temp buffer)
 * @param tmp_d_x       Particle positions (temp buffer)
 * @param tmp_d_u       Particle generalized velocity (temp buffer)
 */
template < coord::cart dir >
__global__
void _bnd_out( int const lim, 
    int * const __restrict__ d_tile_np, int * const __restrict__ d_tile_offset,
    int2 * __restrict__ d_ix, float2 * __restrict__ d_x, float3 * __restrict__ d_u, int * __restrict__ d_idx,
    int * const __restrict__ d_tmp_tile_np, int * const __restrict__ d_tmp_tile_offset, int * const __restrict__ d_tmp_tile_np2, 
    int2 * __restrict__ tmp_d_ix, float2 * __restrict__ tmp_d_x, float3 * __restrict__ tmp_d_u )
{
    auto block = cg::this_thread_block();
    auto warp  = cg::tiled_partition<32>(block);

    const int tid = blockIdx.y * gridDim.x + blockIdx.x;

    unsigned int const np = d_tile_np[ tid ];
    unsigned int const offset =  d_tile_offset[ tid ];
    int2   * __restrict__ ix  = &d_ix[ offset ];
    float2 * __restrict__ x   = &d_x[ offset ];
    float3 * __restrict__ u   = &d_u[ offset ];

    int * __restrict__ idx = &d_idx[ offset ];

    unsigned int const tmp_offset =  d_tmp_tile_offset[ tid ];
    int2   * __restrict__ tmp_ix = &tmp_d_ix[ tmp_offset ];
    float2 * __restrict__ tmp_x  = &tmp_d_x[ tmp_offset ];
    float3 * __restrict__ tmp_u  = &tmp_d_u[ tmp_offset ];


    // Total number of particles leaving
    __shared__ int _nmove;

    // Number of particle leaving to the left neighbour
    __shared__ int _n1;

    _nmove = 0;
    _n1 = 0;

    block.sync();

    int n1 = 0;

    for( int i = block.thread_rank(); i < np; i+= block.num_threads() ) {
        int ipos;
        if ( dir == coord::x ) ipos = ix[i].x;
        if ( dir == coord::y ) ipos = ix[i].y;

        n1 += ( ipos < 0 );

        if ( ( ipos < 0 ) || ( ipos >= lim ) ) {
            int k = atomicAdd( &_nmove, 1 );
            idx[k] = i;
        }
    }

    n1 = cg::reduce( warp, n1, cg::plus<int>());
    if ( warp.thread_rank() == 0 ) atomicAdd( &_n1, n1 );

    block.sync();

    // Number of particles staying in node
    int const _n0 = np - _nmove;

    // Number of particles leaving to the right neighbour
    int const _n2 = _nmove - _n1;

    // Indices for tmp partilce buffer ( 0 - lower, 1 - upper )
    __shared__ int _k[2];
    _k[0] =   0;
    _k[1] = _n1;

    // Index for copying back particles to fill holes
    __shared__ int _c;
    _c = _n0;

    block.sync();

    for( int i = block.thread_rank(); i < _nmove; i+= block.num_threads() ) {
        int src = idx[i];

        int ipos;
        if ( dir == coord::x ) ipos = ix[src].x;
        if ( dir == coord::y ) ipos = ix[src].y;

        int bnd = ( ipos >= lim );

        int k = atomicAdd( & _k[bnd], 1 );

        tmp_ix[k] = ix[src];
        tmp_x[k]  = x[src];
        tmp_u[k]  = u[src];
    }

    block.sync();

    for( int i = block.thread_rank(); i < _nmove; i+= block.num_threads() ) {

        int tgt = idx[i];

        if ( tgt < _n0 ) {
            int c, ipos;

            do {
                c = atomicAdd( &_c, 1 );
                if ( dir == coord::x ) ipos = ix[c].x;
                if ( dir == coord::y ) ipos = ix[c].y;
            } while (( ipos < 0 ) || ( ipos >= lim ));

            ix[ tgt ] = ix[c];
            x[ tgt ]  = x[c];
            u[ tgt ]  = u[c];
        }
    }

    // Store new values on tile information
    if ( block.thread_rank() == 0 ) {
        d_tile_np[ tid ]         = _n0;
        d_tmp_tile_np[ tid ]     = _n1;
        d_tmp_tile_np2[ tid ]    = _n2;
    }

}


/**
 * @brief CUDA kernel for copying in particles that moved out of neighboring
 * tiles into the local tile.
 * 
 * @tparam dir          Direction to check `coord::x` or `coord::y`
 * @param lim           Tile size along chosen direction
 * @param d_tiles       Tile information (main buffer)
 * @param d_ix          Particle cells (main buffer)
 * @param d_x           Particle positions (main buffer)
 * @param d_u           Particle momenta (main buffer)
 * @param tmp_d_tiles   Tile information (temp buffer)
 * @param tmp_d_ix      Particle cells (temp buffer)
 * @param tmp_d_x       Particle positions (temp buffer)
 * @param tmp_d_u       Particle momenta (temp buffer)
 */
template < coord::cart dir > 
__global__
void _bnd_in( int const lim,
    int * const __restrict__ d_tile_np, int * const __restrict__ d_tile_offset,
    int2 * __restrict__ d_ix, float2 * __restrict__ d_x, float3 * __restrict__ d_u,
    int * const __restrict__ d_tmp_tile_np, int * const __restrict__ d_tmp_tile_offset, int * const __restrict__ d_tmp_tile_np2,
    int2 * __restrict__ tmp_d_ix, float2 * __restrict__ tmp_d_x, float3 * __restrict__ tmp_d_u,
    int const periodic )
{

    auto grid  = cg::this_grid(); 
    auto block = cg::this_thread_block();

    const int tid = blockIdx.y * gridDim.x + blockIdx.x;

    unsigned int n0  = d_tile_np[ tid ];
    const int offset =  d_tile_offset[ tid ];
    int2   __restrict__ *ix = &d_ix[ offset ];
    float2 __restrict__ *x  = &d_x[ offset ];
    float3 __restrict__ *u  = &d_u[ offset ];

    // Copy from upper neighbour
    int x_ucoord = blockIdx.x;
    int x_lcoord = blockIdx.x;

    if ( dir == coord::x ) {
        x_lcoord -= 1;
        x_ucoord += 1;
        if ( periodic ) {
            if ( x_lcoord < 0 ) x_lcoord += gridDim.x;
            if ( x_ucoord >= gridDim.x ) x_ucoord -= gridDim.x;
        }
    }

    int y_ucoord = blockIdx.y;
    int y_lcoord = blockIdx.y;

    if ( dir == coord::y ) {
        y_lcoord -= 1;
        y_ucoord += 1;
        if ( periodic ) {
            if ( y_lcoord < 0 ) y_lcoord += gridDim.y;
            if ( y_ucoord >= gridDim.y ) y_ucoord -= gridDim.y;
        }
    }

    if (( x_ucoord < gridDim.x ) && 
        ( y_ucoord < gridDim.y )) {

        int uid = y_ucoord * gridDim.x + x_ucoord;

        unsigned int nu = d_tmp_tile_np[ uid ];
        const int upper_offset =  d_tmp_tile_offset[ uid ];
        int2   __restrict__ *upper_ix = &tmp_d_ix[ upper_offset ];
        float2 __restrict__ *upper_x  = &tmp_d_x[ upper_offset ];
        float3 __restrict__ *upper_u  = &tmp_d_u[ upper_offset ];

        for( int i = block.thread_rank(); i < nu; i+= block.num_threads() ) {
            int2 t = upper_ix[i];

            if ( dir == coord::x ) t.x += lim;
            if ( dir == coord::y ) t.y += lim;

            ix[ n0 + i ] = t;
            x[ n0 + i ]  = upper_x[i];
            u[ n0 + i ]  = upper_u[i];
        }
        n0 += nu;
        
    }

    // Copy from lower neighbour
    if (( x_lcoord >= 0 ) && 
        ( y_lcoord >= 0 )) {

        int lid = y_lcoord * gridDim.x + x_lcoord;;
        
        unsigned int k  = d_tmp_tile_np[ lid ];
        unsigned int nl = d_tmp_tile_np2[ lid ];
        const int lower_offset =  d_tmp_tile_offset[ lid ];
        int2   __restrict__ *lower_ix = &tmp_d_ix[ lower_offset ];
        float2 __restrict__ *lower_x  = &tmp_d_x[ lower_offset ];
        float3 __restrict__ *lower_u  = &tmp_d_u[ lower_offset ];

        for( int i = block.thread_rank(); i < nl; i+= block.num_threads() ) {
            int2 t = lower_ix[k+i];
            
            if ( dir == coord::x ) t.x -= lim;
            if ( dir == coord::y ) t.y -= lim;

            ix[ n0 + i ] = t;
            x[ n0 + i ]  = lower_x[k+i];
            u[ n0 + i ]  = lower_u[k+i];
        }
        n0 += nl;
    }

    if ( block.thread_rank() == 0 ) d_tile_np[ tid ] = n0;
}

/**
 * @brief Moves particles to the correct tiles
 * 
 * Note that particles are only expected to have moved no more than 1 tile
 * in each direction
 * 
 * @param tmp   Temporary buffer to hold particles moving out of tiles. This
 *              buffer *MUST* be big enough to hold all the particles moving
 *              out of the tiles. It's size is not checked.
 */
__host__
void Particles::tile_sort( Particles &tmp ) {
    dim3 grid( ntiles.x, ntiles.y );
    dim3 block( 1024 );

    _bnd_out< coord::x > <<< grid, block >>> ( 
        nx.x, tile_np, tile_offset,
        ix, x, u, idx,
        tmp.tile_np, tmp.tile_offset, tmp.tile_np2,
        tmp.ix, tmp.x, tmp.u
    );

    _bnd_in< coord::x >  <<< grid, block >>> ( 
        nx.x, tile_np, tile_offset, ix, x, u,
        tmp.tile_np, tmp.tile_offset, tmp.tile_np2, tmp.ix, tmp.x, tmp.u,
        periodic.x
    );

    _bnd_out< coord::y > <<< grid, block >>> ( 
        nx.y, tile_np, tile_offset,
        ix, x, u, idx,
        tmp.tile_np, tmp.tile_offset, tmp.tile_np2,
        tmp.ix, tmp.x, tmp.u
     );

    _bnd_in< coord::y >  <<< grid, block >>> ( 
        nx.y, tile_np, tile_offset,
        ix, x, u,
        tmp.tile_np, tmp.tile_offset, tmp.tile_np2, tmp.ix, tmp.x, tmp.u,
        periodic.y
    );

}

__host__
/**
 * @brief Moves particles to the correct tiles
 * 
 * Note that particles are only expected to have moved no more than 1 tile
 * in each direction
 *
 */
void Particles::tile_sort() {

    // Create temporary buffer
    Particles tmp( ntiles, nx, max_np_tile );

    tile_sort( tmp );
}

__global__
void _cell_shift( int * __restrict__ d_tile_np, int * __restrict__ d_tile_offset, 
    int2 * const __restrict__ d_ix,
    int2 const shift )
{
    int const tid = blockIdx.y * gridDim.x + blockIdx.x;

    int const offset = d_tile_offset[ tid ];
    int const np     = d_tile_np[ tid ];
    int2 * const __restrict__ ix = &d_ix[ offset ];

    for( int i = threadIdx.x; i < np; i += blockDim.x) {
        int2 cell = ix[i];
        cell.x += shift.x;
        cell.y += shift.y;
        ix[i] = cell;
    }
}

/**
 * @brief Shifts particle cells by the required amount
 * 
 * Cells are shited by adding the parameter `shift` to the particle cell
 * indexes.
 * 
 * Note that this routine does not check if the particles are still inside the
 * tile.
 * 
 * @param shift     Cell shift in both directions
 */
void Particles::cell_shift( int2 const shift ) {

    dim3 grid( ntiles.x, ntiles.y );
    dim3 block( 1024 );

    _cell_shift <<< grid, block >>> ( tile_np, tile_offset, ix, shift );
}

#define __ULIM __FLT_MAX__

__global__
/**
 * @brief Checks particle buffer data for error
 * 
 * WARNING: This routine is meant for debug only and should not be called 
 *          for production code.
 * 
 * The routine will check for:
 *      1. Invalid cell data (out of tile bounds)
 *      2. Invalid position data (out of [-0.5,0.5[)
 *      3. Invalid momenta (nan, inf or above __ULIM macro value)
 * 
 * If there are any errors found the routine will exit the code.
 * 
 * @param tiles 
 * @param d_ix 
 * @param d_x 
 * @param d_u 
 * @param nx 
 * @param over 
 * @param out 
 */
void _validate( 
    int const * const __restrict__ d_tile_np, 
    int const * const __restrict__ d_tile_offset, 
    int2   const * const __restrict__ d_ix,
    float2 const * const __restrict__ d_x,
    float3 const * const __restrict__ d_u,
    uint2 const nx, int const over, unsigned int * out ) {

    int const tid = blockIdx.y * gridDim.x + blockIdx.x;

    int const offset = d_tile_offset[ tid ];
    int const np     = d_tile_np[ tid ];
    int2   const * const __restrict__ ix = &d_ix[ offset ];
    float2 const * const __restrict__ x  = &d_x[ offset ];
    float3 const * const __restrict__ u  = &d_u[ offset ];

    int2 const lb = make_int2( -over, -over );
    int2 const ub = make_int2( nx.x + over, nx.y + over ); 

    for( int i = threadIdx.x; i < np; i += blockDim.x) {
        int err = 0;

        if ( (ix[i].x < lb.x) || (ix[i].x >= ub.x )) {
            printf("(*error*) Invalid ix[%d].x position (%d), range = [%d,%d[\n", i, ix[i].x, lb.x, ub.x );
            err = 1;
        }
        if ( (ix[i].y < lb.y) || (ix[i].y >= ub.y )) {
            printf("(*error*) Invalid ix[%d].y position (%d), range = [%d,%d[\n", i, ix[i].y, lb.y, ub.y );
            err = 1;
        }

        if ( isnan(u[i].x) || isinf(u[i].x) || fabsf(u[i].x) >= __ULIM ) {
            printf("(*error*) Invalid u[%d].x gen. velocity (%f)\n", i, u[i].x );
            err = 1;
        }

        if ( isnan(u[i].y) || isinf(u[i].y) || fabsf(u[i].x) >= __ULIM ) {
            printf("(*error*) Invalid u[%d].y gen. velocity (%f)\n", i, u[i].y );
            err = 1;
        }

        if ( isnan(u[i].z) || isinf(u[i].z) || fabsf(u[i].x) >= __ULIM ) {
            printf("(*error*) Invalid u[%d].z gen. velocity (%f)\n", i, u[i].z );
            err = 1;
        }

        if ( x[i].x < -0.5f || x[i].x >= 0.5f ) {
            printf("(*error*) Invalid x[%d].x position (%f), range = [-0.5,0.5[\n", i, x[i].x );
            err = 1;
        }
        if ( x[i].y < -0.5f || x[i].y >= 0.5f ) {
            printf("(*error*) Invalid x[%d].y position (%f), range = [-0.5,0.5[\n", i, x[i].y );
            err = 1;
        }

        if ( err ) {
            atomicAdd( out, 1 );
            break;
        }
    }
}


template < coord::cart dir >
__global__
void _validate_dir( 
    int    const * const __restrict__ d_tile_np, 
    int    const * const __restrict__ d_tile_offset, 
    int2   const * const __restrict__ d_ix,
    float2 const * const __restrict__ d_x,
    float3 const * const __restrict__ d_u,
    uint2 const nx, int const over, unsigned int * out ) {

    int const tid = blockIdx.y * gridDim.x + blockIdx.x;

    int const offset = d_tile_offset[ tid ];
    int const np     = d_tile_np[ tid ];
    int2   const * const __restrict__ ix = &d_ix[ offset ];
    float2 const * const __restrict__ x  = &d_x[ offset ];
    float3 const * const __restrict__ u  = &d_u[ offset ];

    int2 const lb = make_int2( -over, -over );
    int2 const ub = make_int2( nx.x + over, nx.y + over ); 

    for( int i = threadIdx.x; i < np; i += blockDim.x) {
        int err = 0;

        if ( dir == coord::x ) {
            if ( (ix[i].x < lb.x) || (ix[i].x >= ub.x )) {
                printf("(*error*) Invalid ix[%d].x position (%d), range = [%d,%d[\n", i, ix[i].x, lb.x, ub.x );
                err = 1;
            }
            if ( x[i].x < -0.5f || x[i].x >= 0.5f ) {
                printf("(*error*) Invalid x[%d].x position (%f), range = [-0.5,0.5[\n", i, x[i].x );
                err = 1;
            }
        }

        if ( dir == coord::x ) {
            if ( (ix[i].y < lb.y) || (ix[i].y >= ub.y )) {
                printf("(*error*) Invalid ix[%d].y position (%d), range = [%d,%d[\n", i, ix[i].y, lb.y, ub.y );
                err = 1;
            }
            if ( x[i].y < -0.5f || x[i].y >= 0.5f ) {
                printf("(*error*) Invalid x[%d].y position (%f), range = [-0.5,0.5[\n", i, x[i].y );
                err = 1;
            }
        }

        if ( isnan(u[i].x) || isinf(u[i].x) || fabsf(u[i].x) >= __ULIM ) {
            printf("(*error*) Invalid u[%d].x gen. velocity (%f)\n", i, u[i].x );
            err = 1;
        }

        if ( isnan(u[i].y) || isinf(u[i].y) || fabsf(u[i].x) >= __ULIM ) {
            printf("(*error*) Invalid u[%d].y gen. velocity (%f)\n", i, u[i].y );
            err = 1;
        }

        if ( isnan(u[i].z) || isinf(u[i].z) || fabsf(u[i].x) >= __ULIM ) {
            printf("(*error*) Invalid u[%d].z gen. velocity (%f)\n", i, u[i].z );
            err = 1;
        }

        if ( err ) {
            atomicAdd( out, 1 );
            break;
        }
    }
}

#undef __ULIM

__host__
/**
 * @brief Validates particle data in buffer
 * 
 * Routine checks for valid positions (both cell index and cell position) and
 * for valid velocities
 * 
 * @param msg       Message to print in case error is found
 * @param over      Amount of extra cells indices beyond limit allowed. Used
 *                  when checking the buffer before tile_sort()
 */
void Particles::validate( std::string msg, int const over ) {

    dim3 grid( ntiles.x, ntiles.y );
    dim3 block( 32 );

    _validate <<< grid, block >>> ( tile_np, tile_offset, ix, x, u, nx, over, _dev_tmp_uint.ptr() );

    unsigned int nerr = _dev_tmp_uint.get();
    if ( nerr > 0 ) {
        std::cerr << "(*error*) " << msg << "\n";
        std::cerr << "(*error*) invalid particle, aborting...\n";
        exit(1);
    }
}

void Particles::validate( std::string msg ) {

    validate( msg, 0 );

}

/**
 * @brief Check if tiles are full
 * 
 * OBSOLETE: To be removed
 * 
 */
void Particles::check_tiles() {

    int * h_tile_np;
    malloc_host( h_tile_np, ntiles.x * ntiles.y );

    devhost_memcpy( h_tile_np, tile_np, ntiles.x * ntiles.y );

    int np = 0;
    int max = 0;

    for( int i = 0; i < ntiles.x * ntiles.y; i++ ) {
        np += h_tile_np[i];
        if ( h_tile_np[i] > max ) max = h_tile_np[i];
    }

    printf("(*info*) #part tile: %g (avg), %d (max), %d (lim)\n", 
        float(np) / (ntiles.x * ntiles.y), max, max_np_tile );

    if ( max >= 0.9 * max_np_tile ) {
        printf("(*critical*) Buffer almost full!\n");
        exit(1);
    }

    free_host( h_tile_np );
}