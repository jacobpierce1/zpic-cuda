#include "hip/hip_runtime.h"
#include <stdio.h>

#include "zpic.h"
#include "timer.cuh"
#include "emf.cuh"
#include "species.cuh"
#include "current.cuh"

#include "simulation.cuh"

/**
 * @brief Tests EM solver and laser injection
 * 
 * Injects a laser pulse and propagates for a given time.
 * 
 */
void test_emf() {
    uint2 ntiles = {16, 16};
    uint2 nx = {16,16};

    float2 box = {25.6, 25.6};
    float dt = 0.07;

    float tmax = 4.0;

    EMF emf( ntiles, nx, box, dt );

    emf.set_moving_window();

/*
    Laser laser;
    laser.start = 16;
    laser.fwhm = 4;
    laser.a0 = 1.0f;
    laser.polarization = 0.f;
    laser.omega0 = 5.0;
*/


    Gaussian laser;
    laser.start = 16;
    laser.fwhm = 4;
    laser.a0 = 1.0f;
    laser.polarization = M_PI_4;
    laser.omega0 = 5.0;

    laser.W0 = 4;
    laser.focus = 12.8;
    laser.axis = 12.8;

    Timer timer;

    timer.start();

    emf.add_laser( laser );

    emf.save( emf::e, fcomp::x );
    emf.save( emf::e, fcomp::y );
    emf.save( emf::e, fcomp::z );

    emf.save( emf::b, fcomp::x );
    emf.save( emf::b, fcomp::y );
    emf.save( emf::b, fcomp::z );

    int iter = 0;

    while( iter * dt < tmax ) {
        emf.advance();
        emf.save( emf::e, fcomp::x );
        emf.save( emf::e, fcomp::y );
        emf.save( emf::e, fcomp::z );

        emf.save( emf::b, fcomp::x );
        emf.save( emf::b, fcomp::y );
        emf.save( emf::b, fcomp::z );

        iter++;
    }

    timer.stop();

    printf("Elapsed time: %.3f ms\n", timer.elapsed());
}

/**
 * @brief Tests the tile sort and current deposit
 * 
 * Creates a sphere of particles and free streams it to check if the tile sort
 * is operating correctly
 * 
 */
void test_sort_deposit() {
    
    std::cout << "Running sort/deposit test...\n";
    
    Timer timer;

    uint2 ntiles = {16, 16};
    uint2 nx = {16,16};

    float2 box = {25.6, 25.6};
    float dt = 0.07;

    Current current( ntiles, nx, box, dt );

    uint2 ppc = {8,8};

    float3 uth = { 0 };
//    float3 uth = {0.1, 0.2, 0.3 };

//    float3 ufl = {0.,    0.,    0.};
//   float3 ufl = {   0.,    0., 1000.};
//    float3 ufl = {1000.,    0.,  1000.};
//    float3 ufl = {   0., 1000.,    1000.};
    float3 ufl = {1000., 1000., 1000.};


    bnd<unsigned int> range;
    range.x = { .lower = 128, .upper = 255 };
    range.y = { .lower = 128, .upper = 255 };

    // auto density = Density::Step( 1.0, 6.4 );
    // auto density =  Density::Slab( 1.0, 9.6, 16.0 );
    auto density = Density::Sphere( 1.0, make_float2(12.8,12.8), 3.2 );

    Species electrons( "electrons", -1, ppc, density, box, ntiles, nx, dt );

    electrons.inject( range );
    electrons.particles->validate( "After injection");
    electrons.set_u( uth, ufl );

    electrons.save_charge();
    current.save( fcomp::x );
    current.save( fcomp::y );
    current.save( fcomp::z );

    timer.start();

    int iter = 0;
    int iter_max = 10;
    while( iter < iter_max ) {
        printf(" i = %3d, t = %g \n", iter, iter * dt );

        current.zero();

        electrons.move( current.J );
        electrons.particles->tile_sort();

        electrons.particles->validate("after tile sort");
        electrons.iter++;

        current.advance();

        electrons.save_charge();
        current.save( fcomp::x );
        current.save( fcomp::y );
        current.save( fcomp::z );

        iter++;
    }

    printf(" i = %3d, t = %g (finished)\n", iter, iter * dt );

    timer.stop();

    printf("Elapsed time: %.3f ms\n", timer.elapsed());
}


void test_move_window() {
    
    std::cout << "Running move window test...\n";
    
    Timer timer;

    uint2 ntiles = {16, 16};
    uint2 nx = {16,16};

    float2 box = {25.6, 25.6};
    float dt = 0.07;

    Simulation sim( ntiles, nx, box, dt );

    // Add particles species
    uint2 ppc  = {8,8};
    float3 ufl = {0., 0., 0.};
    float3 uth = {0., 0., 0.};
    
    auto density = Density::Sphere( 1.0, make_float2(25.6,12.8), 6.4 );
    // auto density = Density::Uniform( 1.0 );

    sim.add_species( "electrons", -1.0f, ppc, density, uth, ufl );

    sim.get_species(0) -> save_charge();

    sim.set_moving_window();

    timer.start();

    int iter_max = 50;
    while( sim.get_iter() < iter_max ) {
        printf(" i = %3d, t = %g \n", sim.get_iter(), sim.get_t() );

        sim.advance();

        sim.get_species(0) -> save_charge();
    }

    printf(" i = %3d, t = %g (finished)\n", sim.get_iter(), sim.get_t() );

    timer.stop();

    printf("Elapsed time: %.3f ms\n", timer.elapsed());
}


void test_weibel() {

    // Create simulation box
    uint2 ntiles = {16, 16};
    uint2 nx = {16,16};
    float2 box = {25.6, 25.6};

    float dt = 0.07;

    Simulation sim( ntiles, nx, box, dt );

    // Add particles species
    uint2 ppc  = {8,8};
    float3 ufl = {0., 0., 0.6};
    float3 uth = {0.1, 0.1, 0.1};

    sim.add_species( "electrons", -1.0f, ppc, Density::Uniform(1.0f), uth, ufl );
    ufl.z = -ufl.z;
    sim.add_species( "positrons", +1.0f, ppc, Density::Uniform(1.0f), uth, ufl );

    // Run simulation
    float const imax = 500;

    printf("Running Weibel test up to n = %g...\n", imax );

    Timer timer;

    timer.start();

    while( sim.get_iter() < imax ) {

        sim.advance();
    }

    timer.stop();

    printf("Simulation complete at i = %d\n", sim.get_iter());
    
    sim.current -> save( fcomp::x );
    sim.current -> save( fcomp::y );
    sim.current -> save( fcomp::z );

    sim.emf -> save( emf::e, fcomp::x );
    sim.emf -> save( emf::e, fcomp::y );
    sim.emf -> save( emf::e, fcomp::z );

    sim.emf -> save( emf::b, fcomp::x );
    sim.emf -> save( emf::b, fcomp::y );
    sim.emf -> save( emf::b, fcomp::z );

    printf("Elapsed time was: %.3f s\n", timer.elapsed( timer::s ));
}

void test_lwfa() {

    // Create simulation box
    uint2 ntiles = {16, 16};
    uint2 nx = {64,16};
    float2 box = {20.48, 25.6};

    float dt = 0.014;

    Simulation sim( ntiles, nx, box, dt );

    // Add particles species
    uint2 ppc  = {8,8};
    float3 ufl = {0., 0., 0.};
    float3 uth = {0., 0., 0.};

    sim.add_species( "electrons", -1.0f, ppc, Density::Step(1.0f,20.48), uth, ufl );

    Gaussian laser;
    laser.start = 17.0;
    laser.fwhm = 2.0;
    laser.a0 = 1.0;
    laser.omega0 = 10.0;
    laser.W0 = 4.0;
    laser.focus = 20.28;
    laser.axis = 12.8;
    laser.polarization = M_PI_2;

    sim.add_laser( laser );

    sim.set_moving_window();

    // Run simulation
    float const tmax = 22.;

    printf("Running LWFA test up to t = %g...\n", tmax );

    Timer timer;

    timer.start();

    while( sim.get_t() < tmax ) {

        sim.advance();
    }

    timer.stop();

    printf("Simulation complete at t = %g\n", sim.get_t());
    
    sim.emf -> save( emf::e, fcomp::x );
    sim.emf -> save( emf::e, fcomp::y );
    sim.emf -> save( emf::e, fcomp::z );

    sim.get_species(0)-> save_phasespace ( 
        phasespace::x,  make_float2( 0., 20.48 ), 1024,
        phasespace::ux, make_float2( -2., 2 ), 512
        );
    sim.get_species(0)->save();

    printf("Elapsed time was: %.3f s\n", timer.elapsed( timer::s ));
}

int main() {

    // test_emf();

    // test_sort_deposit();

    // test_move_window();

    // test_weibel();

    test_lwfa();

    return 0;
}