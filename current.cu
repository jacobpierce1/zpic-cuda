#include "hip/hip_runtime.h"
#include "current.cuh"

#include <iostream>


__host__
/**
 * @brief Construct a new Current:: Current object
 * 
 * @param gnx   Global grid size
 * @param tnx   Tile grid size
 * @param box_  Simulation box dimensions
 * @param dt_   Time step size
 */

/**
 * @brief Construct a new Current:: Current object
 * 
 * @param ntiles    Number of tiles
 * @param nx        Tile grid size
 * @param box       Box size
 * @param dt        Time step
 */
Current::Current( uint2 const ntiles, uint2 const nx, float2 const box,
    float const dt ) : box{box}, dt{dt}
{
    dx.x = box.x / ( nx.x * ntiles.x );
    dx.y = box.y / ( nx.y * ntiles.y );

    // Guard cells (1 below, 2 above)
    // These are required for the Yee solver AND for current deposition
    bnd<unsigned int> gc;
    gc.x = {1,2};
    gc.y = {1,2};

    J = new VectorField( ntiles, nx, gc );

    // Zero initial current
    // This is only relevant for diagnostics, current should always zeroed before deposition
    J -> zero();

    // Set default boundary conditions to periodic
    bc = current::bc_type (current::bc::periodic);

    // Disable filtering by default
    filter = new Filter::None();

    // Reset iteration number
    iter = 0;

    std::cout << "(*info*) Current object initialized." << std::endl;
}

__global__
void _current_bcx(
    float3 * const __restrict__ d_J,
    uint2 const int_nx, uint2 const ext_nx, bnd<unsigned int> gc, 
    uint2 const ntiles, current::bc_type bc )
{
    const int tid = blockIdx.y * ntiles.x + blockIdx.x * (ntiles.x - 1);

    const int tile_off = tid * ext_nx.x * ext_nx.y;
    const int ystride = ext_nx.x;
    const int offset   = gc.x.lower;

    float3 * const __restrict__ J = d_J + tile_off + offset;

    if ( blockIdx.x == 0 ) {
        // Lower boundary
        switch( bc.x.lower ) {
        case( current::bc::reflecting ):
            for( int idx = threadIdx.x; idx < ext_nx.y; idx += blockDim.x ) {
                // j includes the y-stride
                const int j = idx * ystride;

                float jx0 = -J[ -1 + j ].x + J[ 0 + j ].x; 
                float jy1 =  J[ -1 + j ].y + J[ 1 + j ].y;
                float jz1 =  J[ -1 + j ].z + J[ 1 + j ].z;

                J[ -1 + j ].x = J[ 0 + j ].x = jx0;
                J[ -1 + j ].y = J[ 1 + j ].y = jy1;
                J[ -1 + j ].z = J[ 1 + j ].z = jz1;
            }
            break;
        }
    } else {
        // Upper boundary
        switch( bc.x.upper ) {
        case( current::bc::reflecting ):
            for( int idx = threadIdx.x; idx < ext_nx.y; idx += blockDim.x ) {
                int j = idx * ystride;

                float jx0 =  J[ int_nx.x-1 + j ].x - J[ int_nx.x + 0 + j ].x; 
                float jy1 =  J[ int_nx.x-1 + j ].y + J[ int_nx.x + 1 + j ].y;
                float jz1 =  J[ int_nx.x-1 + j ].z + J[ int_nx.x + 1 + j ].z;

                J[ int_nx.x-1 + j ].x = J[ int_nx.x + 0 + j ].x = jx0;
                J[ int_nx.x-1 + j ].y = J[ int_nx.x + 1 + j ].y = jy1;
                J[ int_nx.x-1 + j ].z = J[ int_nx.x + 1 + j ].z = jz1;
            }
            break;
        }
    }
}


__global__
void _current_bcy(
    float3 * const __restrict__ d_J,
    uint2 const int_nx, uint2 const ext_nx, bnd<unsigned int> gc, 
    uint2 const ntiles, current::bc_type bc )
{
    const int tid = blockIdx.y * (ntiles.y - 1) * ntiles.x + blockIdx.x;

    const int tile_off = tid * ext_nx.x * ext_nx.y;
    const int ystride = ext_nx.x;
    const int offset   = gc.y.lower * ystride;

    float3 * const __restrict__ J = d_J + tile_off + offset;

    if ( blockIdx.y == 0 ) {
        // Lower boundary
        switch( bc.y.lower ) {
        case( current::bc::reflecting ):
            for( int idx = threadIdx.x; idx < ext_nx.x; idx += blockDim.x ) {
                int i = idx;

                float jx1 =  J[ i - ystride ].x + J[ i + ystride ].x; 
                float jy0 = -J[ i - ystride ].y + J[ i +       0 ].y;
                float jz1 =  J[ i - ystride ].z + J[ i + ystride ].z;

                J[ i - ystride ].x = J[ i + ystride ].x = jx1;
                J[ i - ystride ].y = J[ i +       0 ].y = jy0;
                J[ i - ystride ].z = J[ i + ystride ].z = jz1;
            }
            break;
        }
    } else {
        // Upper boundary
        switch( bc.y.upper ) {
        case( current::bc::reflecting ):
            for( int idx = threadIdx.x; idx < ext_nx.x; idx += blockDim.x ) {
                int i = idx;

                float jx1 =  J[ i + (int_nx.y-1)*ystride ].x + J[ i + (int_nx.y + 1)*ystride ].x; 
                float jy0 =  J[ i + (int_nx.y-1)*ystride ].y - J[ i + (int_nx.y + 0)*ystride ].y;
                float jz1 =  J[ i + (int_nx.y-1)*ystride ].z + J[ i + (int_nx.y + 1)*ystride ].z;

                J[ i + (int_nx.y-1)*ystride ].x = J[ i + (int_nx.y + 1)*ystride ].x = jx1;
                J[ i + (int_nx.y-1)*ystride ].y = J[ i + (int_nx.y + 0)*ystride ].y = jy0;
                J[ i + (int_nx.y-1)*ystride ].z = J[ i + (int_nx.y + 1)*ystride ].z = jz1;
            }
            break;
        }
    }
}

__host__
/**
 * @brief Processes "physical" boundary conditions
 * 
 */
void Current::process_bc() {

    dim3 block( 64 );

    // x boundaries
    if ( bc.x.lower > current::bc::periodic || bc.x.upper > current::bc::periodic ) {
        dim3 grid( 2, J->ntiles.y );
        _current_bcx <<< grid, block >>> ( J -> d_buffer, J -> nx, J -> ext_nx(), J -> gc, J -> ntiles, bc );
    }

    // y boundaries
    if ( bc.y.lower > current::bc::periodic || bc.y.upper > current::bc::periodic ) {
        dim3 grid( J->ntiles.x, 2 );
        _current_bcy <<< grid, block >>> ( J -> d_buffer, J -> nx, J -> ext_nx(), J -> gc, J -> ntiles, bc );;
    }

}


__host__
/**
 * @brief Advance electric current to next iteration
 * 
 * Adds up current deposited on guard cells and (optionally) applies digital filtering
 * 
 */
void Current::advance() {

    // Add up current deposited on guard cells
    J -> add_from_gc( );
    J -> copy_to_gc( );

    // Do additional bc calculations if needed
    process_bc();

    // Apply filtering
    filter -> apply( *J );

    // Advance iteration count
    iter++;

    // I'm not sure if this should be before or after `iter++`
    // Note that it only affects the axis range on output data
    if ( moving_window.needs_move( iter * dt ) )
        moving_window.advance();
}

__host__
/**
 * @brief Zero electric current values
 * 
 */
void Current::zero() {
    J -> zero();
}

__host__
/**
 * @brief Save electric current data to diagnostic file
 * 
 * @param jc        Current component to save (0, 1 or 2)
 */
void Current::save( fcomp::cart const jc ) {

    char vfname[16];	// Dataset name
    char vflabel[16];	// Dataset label (for plots)

    char comp[] = {'x','y','z'};

    if ( jc < 0 || jc > 2 ) {
        std::cerr << "(*error*) Invalid current component (jc) selected, returning" << std::endl;
        return;
    }

    snprintf(vfname,16,"J%c",comp[jc]);
    snprintf(vflabel,16,"J_%c",comp[jc]);

    zdf::grid_axis axis[2];
    axis[0] = (zdf::grid_axis) {
    	.name = (char *) "x",
    	.min = 0.0 + moving_window.motion(),
    	.max = box.x,
    	.label = (char *) "x",
    	.units = (char *) "c/\\omega_n"
    };

    axis[1] = (zdf::grid_axis) {
        .name = (char *) "y",
    	.min = 0.0 + moving_window.motion(),
    	.max = box.y,
    	.label = (char *) "y",
    	.units = (char *) "c/\\omega_n"
    };

    zdf::grid_info info = {
        .name = vfname,
    	.ndims = 2,
    	.label = vflabel,
    	.units = (char *) "e \\omega_n^2 / c",
    	.axis = axis
    };

    info.count[0] = J -> ntiles.x * J -> nx.x;
    info.count[1] = J -> ntiles.y * J -> nx.y;

    zdf::iteration iteration = {
    	.n = iter,
    	.t = iter * dt,
    	.time_units = (char *) "1/\\omega_n"
    };

    J -> save( jc, info, iteration, "CURRENT" );
}